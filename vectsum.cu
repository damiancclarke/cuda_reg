#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


int main(void) {
  int *a,*b,*c;
  int *dev_a, *dev_b, *dev_c;
  int size = N * sizeof(int);

  hipMalloc( (void**)&dev_a, size);
  hipMalloc( (void**)&dev_b, size);
  hipMalloc( (void**)&dev_c, size);

  a = (int*)malloc(size);
  b = (int*)malloc(size);
  c = (int*)malloc(size);

  random_ints(a, N);
  random_ints(b, N);

  hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
  hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

  
  add<<<N,1>>> (dev_a,dev_b,dev_c);

  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost );

  free(a); free(b); free(c);
  hipFree( dev_a );
  hipFree( dev_b );
  hipFree( dev_c );

  return 0;
}

void random_ints(int* a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
    a[i] = rand();
}
